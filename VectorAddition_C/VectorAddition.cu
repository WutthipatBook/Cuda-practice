// VectorAddition.cu
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512

__global__ void vectorAdd(const float *A, const float *B, float *C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int size = N * sizeof(float);
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    h_A = (float *)malloc(size);
    h_B = (float *)malloc(size);
    h_C = (float *)malloc(size);

    // Initialize input vectors
    for (int i = 0; i < N; ++i) {
        h_A[i] = i * 1.0f;
        h_B[i] = i * 2.0f;
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy input vectors to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print a few results
    for (int i = 0; i < 10; ++i) {
        printf("C[%d] = %f\n", i, h_C[i]);
    }

    // Cleanup
    free(h_A); free(h_B); free(h_C);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    return 0;
}