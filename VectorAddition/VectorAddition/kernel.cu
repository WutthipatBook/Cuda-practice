﻿
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(float);

    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];

    // Initialize vectors
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Check results
    for(int i = 0; i < 20; i++) {
        std::cout << h_A[i] << " + " << h_B[i] << " = " << h_C[i] << "\n";
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;

    return 0;
}